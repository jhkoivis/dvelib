
#include <stdio.h>
#include <hip/hip_runtime.h>

#define	SIM_THREADS		10		// how many simultaneus threads
#define N	100					// number of variables in a vector


// this function does absolutely nothing, but runs on multiple cores
__global__ void dummyFunct(void)
{
	int i;
	int a = 0;

	// this loop will do sequences:
	//	i = 0, 10, 20, ...
	//  i = 1, 11, 21, ...
	//  i = 2, 12, 22, ...
	//  ...
	//  i = 9, 19, 29, ...
	//
	// assuming SIM_THREADS = 10

	for (	i = threadIdx.x;	// start from i = thread ID
			i < N; 				// stop if all i's are done
			i += SIM_THREADS)	// skip number of threads
		a += 1;
}

int main(void)
{
	dummyFunct<<<1,SIM_THREADS>>>();
}


