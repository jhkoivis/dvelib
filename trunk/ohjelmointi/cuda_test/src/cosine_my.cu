/* Cuda GPU Based Program that use GPU processor for finding cosine of numbers */

/* --------------------------- header secton ----------------------------*/
#include<stdio.h>
#include<hip/hip_runtime.h>

#define COS_THREAD_CNT 2
#define N 10

/* --------------------------- target code ------------------------------*/
struct cosParams {
	float *arg;
	float *res;
	int n;
	//int *threadIdx;
};

__global__ void cos_main(struct cosParams parms)
{
	int i;
	for (i = threadIdx.x; i < parms.n; i += COS_THREAD_CNT)
	{
		parms.res[i] = __cosf(parms.arg[i] );
		//parms.threadIdx[0] = (int)threadIdx.x;
	}
}

/* --------------------------- host code ------------------------------*/
int main (int argc, char *argv[])
{
	int 			i = 0;
	hipError_t 	cudaStat;
	//int*			cosThreadIdx = 0;
	float* 			cosRes = 0;
	float*			cosArg = 0;
	//int*			threadIdx = (int *) malloc(N*sizeof(threadIdx));
	float* 			arg = (float *) malloc(N*sizeof(arg[0]));
	float*			res = (float *) malloc(N*sizeof(res[0]));
	struct cosParams funcParams;


	/* ... fill arguments array "arg" .... */
	for(i=0; i < N; i++ ){
		arg[i] = (float)i;
	}

	cudaStat = hipMalloc ((void **)&cosArg, 	N * sizeof(cosArg[0]));
	cudaStat = hipMalloc ((void **)&cosRes, 	N * sizeof(cosRes[0]));
	//cudaStat = cudaMalloc ((void **)&threadIdx, N * sizeof(threadIdx[0]));
	cudaStat = hipMemcpy (	cosArg,
							arg,
							N * sizeof(arg[0]),
							hipMemcpyHostToDevice);

	funcParams.res = cosRes;
	funcParams.arg = cosArg;
	funcParams.n = N;
	//funcParams.threadIdx = cosThreadIdx;
	cos_main<<<1,COS_THREAD_CNT>>>(funcParams);

	cudaStat = hipMemcpy(	res,
							cosRes,
							N * sizeof(cosRes[0]),
							hipMemcpyDeviceToHost);

	//cudaStat = cudaMemcpy(	threadIdx,
	//						cosThreadIdx,
	//						N * sizeof(cosThreadIdx[0]),
	//						cudaMemcpyDeviceToHost);

	for(i=0; i < N; i++ )
	{
		printf("%d: cosf(%f) = %f\n", arg[i], res[i]); //, threadIdx[i]);
	}
}

/* nvcc cosine.cu -use_fast_math */
