#include "hip/hip_runtime.h"

#include "sum.h"

__global__ void convolutionKernel(	float *pSrcImg,
									size_t srcPitch,
									float *pDefImg,
									size_t defPitch,
									float *pSumImg,
									size_t sumPitch)
{
	int 	x,
			y;
	x = threadIdx.x + blockDim.x * blockIdx.x;
	y = threadIdx.y + blockDim.y * blockIdx.y;

	pSumImg[x + y*sumPitch] = 	pDefImg[x + y*defPitch] +
								pSrcImg[x + y*srcPitch];
}

void pce(	hipError_t *pCudaError,
			int lineNumber)
{
	if (*pCudaError) printf( 	"hipError_t at line %d:\n  %s\n",
								lineNumber,
								hipGetErrorString(*pCudaError));
}

int copyHtoD(struct	CudaMemPointers	*cMem)
{
	hipError_t
			hipError_t;

	// memory for the device
	hipError_t = hipMallocPitch((void **) &(cMem->pSrcImg_device),
								&(cMem->srcImgPitch),
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemset2D (	cMem->pSrcImg_device,
								cMem->srcImgPitch,
								0,
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMallocPitch((void **) &(cMem->pDefImg_device),
								&(cMem->defImgPitch),
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemset2D (	cMem->pDefImg_device,
								cMem->defImgPitch,
								0,
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMallocPitch((void **) &(cMem->pSumImg_device),
								&(cMem->sumImgPitch),
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemset2D (	cMem->pSumImg_device,
								cMem->sumImgPitch,
								0,
								cMem->xSize*sizeof(float),
								cMem->ySize);
	pce(&hipError_t, __LINE__);

	//copy memory from host to device
	hipError_t = hipMemcpy2D(	cMem->pSrcImg_device,
								cMem->srcImgPitch,
								cMem->pSrcImg_host,
								cMem->xSize*sizeof(float),
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMemcpy2D(	cMem->pDefImg_device,
								cMem->defImgPitch,
								cMem->pDefImg_host,
								cMem->xSize*sizeof(float),
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMemcpy2D(	cMem->pSumImg_device,
								cMem->sumImgPitch,
								cMem->pSumImg_host,
								cMem->xSize*sizeof(float),
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	return 0;
}

int copyDtoH(	struct CudaMemPointers 	*cMem)
{
	hipError_t
			hipError_t;

	hipError_t = hipMemcpy2D(	cMem->pSrcImg_host,
								cMem->xSize*sizeof(float),
								cMem->pSrcImg_device,
								cMem->srcImgPitch,
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMemcpy2D(	cMem->pDefImg_host,
								cMem->xSize*sizeof(float),
								cMem->pDefImg_device,
								cMem->defImgPitch,
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);

	hipError_t = hipMemcpy2D(	cMem->pSumImg_host,
								cMem->xSize*sizeof(float),
								cMem->pSumImg_device,
								cMem->defImgPitch,
								cMem->xSize*sizeof(float),
								cMem->ySize,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);

	return 0;
}

float sum(float *pSrcImg,
		float *pDefImg,
		float *pSumImg,
		int xSize,
		int ySize) {

	CudaMemPointers	cMem;

	//int		i,
	//		j,
	//		testFailed = 0;

	hipError_t
			hipError_t;

	unsigned int timer = 0;
	float compute_time;

	/*
	for (i = 0; i < imDimX; i++)
	{
		for (j=0; j < imDimY; j++)
		{
			printf(	"%d, %d: %f %f %f\n", i,j,
				pSrcImg[i+j*imDimX],
				pDefImg[i+j*imDimX],
				pSumImg[i+j*imDimX]);
		}
	}
	*/

	cMem.pSrcImg_host = pSrcImg;
	cMem.pDefImg_host = pDefImg;
	cMem.pSumImg_host = pSumImg;
	cMem.xSize = xSize;
	cMem.ySize = ySize;

	hipSetDevice(0);

	// copy data from host to device
	// fixme: this also reserves memory -> make separate function
	copyHtoD(&cMem);

	// create grid (this is c++ ?)
	dim3 block(BLOCK_X,BLOCK_Y);
	dim3 grid(cMem.xSize/BLOCK_X,cMem.ySize/BLOCK_Y);


	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));
	convolutionKernel<<< grid, block >>>(	cMem.pSrcImg_device,
											cMem.srcImgPitch/sizeof(float),
											cMem.pDefImg_device,
											cMem.defImgPitch/sizeof(float),
											cMem.pSumImg_device,
											cMem.sumImgPitch/sizeof(float)); // IMPORTANT SIZEOF
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);
	hipDeviceSynchronize();
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);

	CUT_SAFE_CALL(cutStopTimer(timer));
	compute_time = cutGetTimerValue(timer);
	cutDeleteTimer(timer);
	//printf("kernel execution time : %f (ms)\n", compute_time);

	// copy results back
	copyDtoH(&cMem);

	/*
	for (i = 0; i < imDimX; i++)
	{
		for (j=0; j < imDimY; j++)
		{
			testValue = pSrcImg[i + j*imDimX] + pDefImg[i + j*imDimX];
			testValue = testValue - pSumImg[i + j*imDimX];
			testValue = testValue * testValue;
			if (testValue > 0.001 )
				{
					printf("%d, %d: %f", i,j,pSrcImg[i+j*imDimX]);
					//printf(" %f\n", (RANDNUM));
					testFailed = 1;
				}
			else
			{
				printf(	"%d, %d: %f %f %f\n", i,j,
						pSrcImg[i+j*imDimX],
						pDefImg[i+j*imDimX],
						pSumImg[i+j*imDimX]);

			}
		}
	}
	if (testFailed != 0)
	{
		printf("Test failed\n");
	}
	else
	{
		printf("Test passed\n");
	}
	*/

	// free data
	//hipError_t = hipFree(pSrcImg_device);
	//pce(&hipError_t, __LINE__);
	//hipError_t = hipFree(pDefImg_device);
	//pce(&hipError_t, __LINE__);
	//hipError_t = hipHostFree(	(void **)&pSrcImg);
	//pce(&hipError_t, __LINE__);

	return compute_time;

}
