#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//#include <cutil.h>

#define BLOCK_X 16
#define BLOCK_Y 16


__global__ void convolutionKernel(	float *pSrcImg)
{
	int 	x,
			y;
	x = threadIdx.x + blockDim.x * blockIdx.x;
	y = threadIdx.y + blockDim.y * blockIdx.y;
	pSrcImg[x + y*blockDim.x] = 1;
}

void pce(	hipError_t *pCudaError,
			int lineNumber)
{
	if (*pCudaError) printf( 	"hipError_t at line %d:\n  %s\n",
								lineNumber,
								hipGetErrorString(*pCudaError));
}

int main(int argc, char **argv) {

	float 	*pSrcImg,
			*pSrcImg_device;
	//size_t
	//		srcImgPitch;

			//*pRefImg,
			//*pRefImg_device,
			//refImgPitch;

	int		imDimX = 16,
			imDimY = 16,
			i,
			testFailed = 0;

	hipError_t
			hipError_t;

	hipSetDevice(0);

	// memory for the device
	hipError_t = hipMalloc(	(void **) &pSrcImg_device,
							imDimY*imDimX*sizeof(float));
	pce(&hipError_t, __LINE__);

	// memory for the host
	pSrcImg = (float *)calloc(imDimY*imDimX, sizeof(float));
	//hipError_t = hipHostMalloc(	(void **)&pSrcImg,
	//							(size_t)imDimX*imDimY*sizeof(float));
	//pce(&hipError_t, __LINE__);

	//copy memory from host to device
	hipError_t = hipMemcpy(pSrcImg_device,
							pSrcImg,
							imDimY*imDimX*sizeof(float),
							hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	// create grid (this is c++ ?)
	dim3 block(	BLOCK_X,1);
	dim3 grid(imDimX*imDimY/BLOCK_X,1);

	convolutionKernel<<< grid, block >>>(pSrcImg_device);
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);
	hipDeviceSynchronize();
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);

	// copy results back
	hipError_t = hipMemcpy(	pSrcImg,
							pSrcImg_device,
							imDimY*imDimX*sizeof(float),
							hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);
	for (i = 0; i < imDimX*imDimY; i++)
	{
		if ((int)pSrcImg[i] != 1) testFailed = 1;
	}
	if (testFailed != 0)
	{
		printf("Test failed\n");
	}
	else
	{
		printf("Test passed\n");
	}

	// free data
	//hipError_t = hipFree(		(void **)&pSrcImg_device);
	//pce(&hipError_t, __LINE__);
	//hipError_t = hipHostFree(	(void **)&pSrcImg);
	//pce(&hipError_t, __LINE__);

	return hipError_t;

}
