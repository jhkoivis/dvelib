/* Cuda GPU Based Program that use GPU processor for finding cosine of numbers */

/* --------------------------- header secton ----------------------------*/
#include<stdio.h>
#include<hip/hip_runtime.h>

#define COS_THREAD_CNT 10
#define N 10

/* --------------------------- target code ------------------------------*/
struct cosParams {
	float *arg;
	float *res;
	int n;
};

struct	myParams {
	int	*number;
	int n;
};


__global__ void myFunction(struct myParams parms)
{
	int i;
	for (i = threadIdx.x; i < parms.n; i += COS_THREAD_CNT)
	{
		parms.number[0] = i;
	}
}

__global__ void cos_main(struct cosParams parms)
{
	int i;
	for (i = threadIdx.x; i < parms.n; i += COS_THREAD_CNT) {
		parms.res[i] = __cosf(parms.arg[i] );
	}
}

/* --------------------------- host code ------------------------------*/
int main (int argc, char *argv[])
{
	int i = 0;
	hipError_t cudaStat;
	float* cosRes = 0;
	float* cosArg = 0;
	float* arg = (float *) malloc(N*sizeof(arg[0]));
	float* res = (float *) malloc(N*sizeof(res[0]));
	struct cosParams funcParams;

	struct myParams myFucnParams;

	/* ... fill arguments array "arg" .... */
	for(i=0; i < N; i++ ){
		arg[i] = (float)i;
	}

	cudaStat = hipMalloc ((void **)&cosArg, N * sizeof(cosArg[0]));
	if( cudaStat )
		printf(" value = %d : Memory Allocation on GPU Device failed\n", cudaStat);

	cudaStat = hipMalloc ((void **)&cosRes, N * sizeof(cosRes[0]));
	if( cudaStat )
		printf(" value = %d : Memory Allocation on GPU Device failed\n", cudaStat);

	cudaStat = hipMemcpy (cosArg, arg, N * sizeof(arg[0]), hipMemcpyHostToDevice);
	if( cudaStat )
		printf(" Memory Copy from Host to Device failed.\n", cudaStat);

	funcParams.res = cosRes;
	funcParams.arg = cosArg;
	funcParams.n = N;
	cos_main<<<1,COS_THREAD_CNT>>>(funcParams);

	cudaStat = hipMemcpy (res, cosRes, N * sizeof(cosRes[0]), hipMemcpyDeviceToHost);
	if( cudaStat )
		printf(" Memory Copy from Device to Host failed.\n" , cudaStat);

	for(i=0; i < N; i++ ){
		printf("cosf(%f) = %f \n", arg[i], res[i] );
	}
}

/* nvcc cosine.cu -use_fast_math */
