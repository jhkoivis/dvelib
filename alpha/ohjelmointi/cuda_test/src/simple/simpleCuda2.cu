
#include <stdio.h>
#include <hip/hip_runtime.h>

#define	SIM_THREADS		10		// how many simultaneus threads
#define N	20 					// number of variables in a vector


// this function returns a result
__global__ void dummyFunct(float *pResult)
{
	int i;
	float previous = 0.0;
	pResult[0] = 0.0;
	// this loop will do sequences:
	//	i = 0, 10, 20, ...
	//  i = 1, 11, 21, ...
	//  i = 2, 12, 22, ...
	//  ...
	//  i = 9, 19, 29, ...
	//
	// assuming SIM_THREADS = 10

	for (	i = threadIdx.x;	// start from i = thread ID
			i < N; 				// stop if all i's are done
			i += SIM_THREADS)	// skip number of threads
		pResult[i] = previous + i;
		previous = pResult[i];
}

int main(void)
{
	float *pHostResult;
	float *pCudaResult = 0;
	int i;

	// reserve memory in host system
	pHostResult = (float *) malloc(N*sizeof(pHostResult[0]));

	// reserve memory in cuda
	hipMalloc((void **) &pCudaResult, N*sizeof(pCudaResult[0]));

	dummyFunct<<<1,SIM_THREADS>>>(pCudaResult);

	// copy result from cuda to host
	hipMemcpy(	pHostResult, 				// destination
				pCudaResult, 				// source
				N*sizeof(pCudaResult[0]),	// amount to copy
				hipMemcpyDeviceToHost);	// type: device -> host

	for (i = 0; i < N; i++)
		printf("%f\n", pHostResult[i]);
}


