
#include <stdio.h>
#include <hip/hip_runtime.h>

#define	SIM_THREADS		10		// how many simultaneus threads
#define N	20 					// number of variables in a vector


// this function returns a result
__global__ void cudaFunct(float *pArgument, float *pResult)
{
	int i;
	// this loop will do sequences:
	//	i = 0, 10, 20, ...
	//  i = 1, 11, 21, ...
	//  i = 2, 12, 22, ...
	//  ...
	//  i = 9, 19, 29, ...
	//
	// assuming SIM_THREADS = 10

	for (	i = threadIdx.x;	// start from i = thread ID
			i < N; 				// stop if all i's are done
			i += SIM_THREADS)	// skip number of threads
		pResult[i] = pArgument[i] -pArgument[i-1];
}

int main(void)
{
	float *pHostArgument;
	float *pCudaArgument = 0;
	float *pHostResult;
	float *pCudaResult = 0;
	int i;

	// reserve memory in host system
	pHostArgument = (float *)malloc(N*sizeof(pHostArgument[0]));
	pHostResult = (float *) malloc(N*sizeof(pHostResult[0]));

	// reserve memory in cuda
	hipMalloc((void **) &pCudaArgument, N*sizeof(pCudaResult[0]));
	hipMalloc((void **) &pCudaResult, N*sizeof(pCudaResult[0]));

	// initialize argument
	for (i = 0; i < N; i++) pHostArgument[i] = float(i);

	// copy argument from host to cuda
	hipMemcpy(	pCudaArgument, 				// destination
				pHostArgument, 				// source
				N*sizeof(pCudaResult[0]),	// amount to copy
				hipMemcpyHostToDevice);	// type: host -> device

	// execute in cuda
	cudaFunct<<<1,SIM_THREADS>>>(pCudaArgument, pCudaResult);

	// copy result from cuda to host
	hipMemcpy(	pHostResult, 				// destination
				pCudaResult, 				// source
				N*sizeof(pCudaResult[0]),	// amount to copy
				hipMemcpyDeviceToHost);	// type: device -> host

	for (i = 0; i < N; i++)
		printf("%f\n", pHostResult[i]);
}

