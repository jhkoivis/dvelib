#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//#include <cutil.h>

#define BLOCK_X 16
#define BLOCK_Y 16


__global__ void convolutionKernel(	float *pSrcImg, size_t pitch)
{
	int 	x,
			y;
	x = threadIdx.x + blockDim.x * blockIdx.x;
	y = threadIdx.y + blockDim.y * blockIdx.y;
	pSrcImg[x + y*pitch] = 1;
}

void pce(	hipError_t *pCudaError,
			int lineNumber)
{
	if (*pCudaError) printf( 	"hipError_t at line %d:\n  %s\n",
								lineNumber,
								hipGetErrorString(*pCudaError));
}

int main(int argc, char **argv) {

	float 	*pSrcImg,
			*pSrcImg_device;
			//*pRefImg,
			//*pRefImg_device,
	size_t
			srcImgPitch;
			//refImgPitch;

	int		imDimX = 64,
			imDimY = 64,
			i,
			j,
			testFailed = 0;

	hipError_t
			hipError_t;

	hipSetDevice(0);

	// memory for the device
	hipError_t = hipMallocPitch(	(void **) &pSrcImg_device,
									&srcImgPitch,
									imDimX*sizeof(float),
									imDimY);
	pce(&hipError_t, __LINE__);
	printf("pitch: %d\n", (int)srcImgPitch);

	// memory for the host
	pSrcImg = (float *)calloc(imDimY*imDimX, sizeof(float));
	//hipError_t = hipHostMalloc(	(void **)&pSrcImg,
	//							(size_t)imDimX*imDimY*sizeof(float));
	//pce(&hipError_t, __LINE__);

	//copy memory from host to device
	hipError_t = hipMemcpy2D(	pSrcImg_device,
								srcImgPitch,
								pSrcImg,
								imDimX*sizeof(float),
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	// create grid (this is c++ ?)
	dim3 block(BLOCK_X,BLOCK_Y);
	dim3 grid(imDimX/BLOCK_X,imDimY/BLOCK_Y);

	convolutionKernel<<< grid, block >>>(	pSrcImg_device,
											srcImgPitch/sizeof(float)); // IMPORTANT SIZEOF
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);
	hipDeviceSynchronize();
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);

	// copy results back
	hipError_t = hipMemcpy2D(	pSrcImg,
								imDimX*sizeof(float),
								pSrcImg_device,
								srcImgPitch,
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);
	for (i = 0; i < imDimX; i++)
	{
		for (j=0; j < imDimY; j++)
		{
			if ((int)pSrcImg[i+j*imDimY] != 1)
				{
					//printf("%d, %d: %f\n", i,j,pSrcImg[i+j*imDimY]);
					testFailed = 1;
				}
			else
			{
				//printf("%d, %d: %f\n", i,j,pSrcImg[i+j*imDimY]);
			}
		}
	}
	if (testFailed != 0)
	{
		printf("Test failed\n");
	}
	else
	{
		printf("Test passed\n");
	}

	// free data
	//hipError_t = hipFree(		(void **)&pSrcImg_device);
	//pce(&hipError_t, __LINE__);
	//hipError_t = hipHostFree(	(void **)&pSrcImg);
	//pce(&hipError_t, __LINE__);

	return hipError_t;

}
