#include "hip/hip_runtime.h"

#include "convolution.h"

__global__ void convolutionKernel(	float *pSrcImg,
									size_t srcPitch,
									float *pDefImg,
									size_t defPitch)
{
	int 	x,
			y;
	x = threadIdx.x + blockDim.x * blockIdx.x;
	y = threadIdx.y + blockDim.y * blockIdx.y;
	pDefImg[x + y*defPitch] = pSrcImg[x + y*srcPitch];
}

void pce(	hipError_t *pCudaError,
			int lineNumber)
{
	if (*pCudaError) printf( 	"hipError_t at line %d:\n  %s\n",
								lineNumber,
								hipGetErrorString(*pCudaError));
}

int main(void) {

	float 	*pSrcImg,
			*pSrcImg_device,
			*pDefImg,
			*pDefImg_device,
			testValue;
	size_t
			srcImgPitch,
			defImgPitch;

	int		imDimX = 2048,
			imDimY = 2048,
			i,
			j,
			testFailed = 0;

	hipError_t
			hipError_t;

	unsigned int timer = 0;
	float compute_time;

	hipSetDevice(0);

	// memory for the device
	hipError_t = hipMallocPitch(	(void **) &pSrcImg_device,
									&srcImgPitch,
									imDimX*sizeof(float),
									imDimY);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemset2D (	pSrcImg_device,
								srcImgPitch,
								0,
								imDimX*sizeof(float),
								imDimY);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMallocPitch(	(void **) &pDefImg_device,
									&defImgPitch,
									imDimX*sizeof(float),
									imDimY);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemset2D (	pDefImg_device,
								defImgPitch,
								0,
								imDimX*sizeof(float),
								imDimY);
	pce(&hipError_t, __LINE__);


	// memory for the host
	pSrcImg = (float *)calloc(imDimY*imDimX, sizeof(float));
	pDefImg = (float *)calloc(imDimY*imDimX, sizeof(float));

	for (i = 0; i < imDimX; i++)
	{
		for (j = 0; j < imDimY; j++)
		{
			pSrcImg[i + j*imDimX] = (RANDNUM);
		}
	}

	//hipError_t = hipHostMalloc(	(void **)&pSrcImg,
	//							(size_t)imDimX*imDimY*sizeof(float));
	//pce(&hipError_t, __LINE__);

	//copy memory from host to device
	hipError_t = hipMemcpy2D(	pSrcImg_device,
								srcImgPitch,
								pSrcImg,
								imDimX*sizeof(float),
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemcpy2D(	pDefImg_device,
								defImgPitch,
								pDefImg,
								imDimX*sizeof(float),
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyHostToDevice);
	pce(&hipError_t, __LINE__);

	// create grid (this is c++ ?)
	dim3 block(BLOCK_X,BLOCK_Y);
	dim3 grid(imDimX/BLOCK_X,imDimY/BLOCK_Y);


	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));
	convolutionKernel<<< grid, block >>>(	pSrcImg_device,
											srcImgPitch/sizeof(float),
											pDefImg_device,
											defImgPitch/sizeof(float)); // IMPORTANT SIZEOF
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);
	hipDeviceSynchronize();
	hipError_t = hipGetLastError();
	pce(&hipError_t, __LINE__);

	CUT_SAFE_CALL(cutStopTimer(timer));
	compute_time = cutGetTimerValue(timer);
	cutDeleteTimer(timer);
	printf("kernel execution time : %f (ms)\n", compute_time);




	// copy results back
	hipError_t = hipMemcpy2D(	pSrcImg,
								imDimX*sizeof(float),
								pSrcImg_device,
								srcImgPitch,
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);
	hipError_t = hipMemcpy2D(	pDefImg,
								imDimX*sizeof(float),
								pDefImg_device,
								defImgPitch,
								imDimX*sizeof(float),
								imDimY,
								hipMemcpyDeviceToHost);
	pce(&hipError_t, __LINE__);
	for (i = 0; i < imDimX; i++)
	{
		for (j=0; j < imDimY; j++)
		{
			testValue = (pSrcImg[i+j*imDimX] - (RANDNUM))* (pSrcImg[i+j*imDimX] - (RANDNUM));
			if (testValue > 0.001 )
				{
					//printf("%d, %d: %f", i,j,pSrcImg[i+j*imDimX]);
					//printf(" %f\n", (RANDNUM));
					testFailed = 1;
				}
			else
			{
				//printf("%d, %d: %f\n", i,j,pSrcImg[i+j*imDimY]);
			}
		}
	}
	if (testFailed != 0)
	{
		printf("Test failed\n");
	}
	else
	{
		printf("Test passed\n");
	}

	// free data
	hipError_t = hipFree(pSrcImg_device);
	pce(&hipError_t, __LINE__);
	hipError_t = hipFree(pDefImg_device);
	pce(&hipError_t, __LINE__);
	//hipError_t = hipHostFree(	(void **)&pSrcImg);
	//pce(&hipError_t, __LINE__);

	return hipError_t;

}
